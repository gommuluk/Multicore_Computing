#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define SPHERES 20

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

struct Sphere {
	float   r, b, g;
	float   radius;
	float   x, y, z;
};

/* Added __global__ variable to run kernel function in GPU */
__global__ void kernel(const Sphere* s, unsigned char* ptr)
{
	/* Each kernel function uses thread/block index and block dimension to determine a unique number to process a particular pixel (x, y) */
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= DIM || y >= DIM) return;

	int offset = x + y * DIM;
	float ox = (x - DIM / 2);
	float oy = (y - DIM / 2);

	float r = 0, g = 0, b = 0;
	float   maxz = -INF;

	for (int i = 0; i<SPHERES; i++) {
		float   n;
		float   t;

		/* Moved the 'hit' function from the 'Sphere' structure of the existing code */
		float dx = ox - s[i].x;
		float dy = oy - s[i].y;

		if (dx*dx + dy*dy < s[i].radius*s[i].radius) {
			float dz = sqrtf(s[i].radius*s[i].radius - dx*dx - dy*dy);
			n = dz / sqrtf(s[i].radius * s[i].radius);
			t = dz + s[i].z;
		}
		else t = -INF;

		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		}
	}

	ptr[offset * 4 + 0] = (int)(r * 255);
	ptr[offset * 4 + 1] = (int)(g * 255);
	ptr[offset * 4 + 2] = (int)(b * 255);
	ptr[offset * 4 + 3] = 255;
}

void ppm_write(unsigned char* bitmap, int xdim, int ydim, FILE* fp)
{
	int i, x, y;
	fprintf(fp, "P3\n");
	fprintf(fp, "%d %d\n", xdim, ydim);
	fprintf(fp, "255\n");
	for (y = 0; y<ydim; y++) {
		for (x = 0; x<xdim; x++) {
			i = x + y*xdim;
			fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
		}
		fprintf(fp, "\n");
	}
}

int main(int argc, char* argv[])
{
	int x, y;
	unsigned char* bitmap;
	hipEvent_t start, stop;  // for time measurement
	float timeDiff;

	/* time variables event create */
	hipEventCreate(&start);
	hipEventCreate(&stop);
	srand(time(NULL));

	if (argc != 2) {
		printf("> a.out [filename.ppm]\n");
		exit(0);
	}
	FILE* fp = fopen(argv[1], "w");

	Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
	for (int i = 0; i<SPHERES; i++) {
		temp_s[i].r = rnd(1.0f);
		temp_s[i].g = rnd(1.0f);
		temp_s[i].b = rnd(1.0f);
		temp_s[i].x = rnd(2000.0f) - 1000;
		temp_s[i].y = rnd(2000.0f) - 1000;
		temp_s[i].z = rnd(2000.0f) - 1000;
		temp_s[i].radius = rnd(200.0f) + 40;
	}

	bitmap = (unsigned char*)malloc(sizeof(unsigned char)*DIM*DIM * 4);

	/* device_s and device_bitmap is to be assigned to device */
	Sphere *device_s;
	unsigned char* device_bitmap;

	/* Allocate space on GPU to copy the temp_s */
	hipMalloc((void**)&device_s, sizeof(Sphere)*SPHERES);
	hipMalloc((void**)&device_bitmap, sizeof(unsigned char)*DIM*DIM * 4);

	/* Copy temp_s to device_s to run the function in GPU */
	hipMemcpy(device_s, temp_s, sizeof(Sphere)*SPHERES, hipMemcpyHostToDevice);

	/* Start the recoding */
	hipEventRecord(start, 0);

	/* 768 thread per block */
	dim3 dimBlock(32, 24);
	/* (Dimension/blockDimension) block per grid */
	dim3 dimGrid(DIM / dimBlock.x, DIM / dimBlock.y);

	kernel << <dimGrid, dimBlock >> > (device_s, device_bitmap);
	hipDeviceSynchronize();

	/* End the recording */
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	/* Copy the result stored in the device back to the host */
	hipMemcpy(bitmap, device_bitmap, sizeof(unsigned char)*DIM*DIM * 4, hipMemcpyDeviceToHost);

	/* Execution time checking */
	hipEventElapsedTime(&timeDiff, start, stop);
	printf("CUDA ray tracing: %f sec \n", timeDiff / CLOCKS_PER_SEC);

	ppm_write(bitmap, DIM, DIM, fp);
	printf("[%s] was generated. \n", argv[1]);

	fclose(fp);
	free(bitmap);
	free(temp_s);

	hipFree(device_s);
	hipFree(device_bitmap);

	return 0;
}